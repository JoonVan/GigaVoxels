#include "hip/hip_runtime.h"
/*
 * GigaVoxels is a ray-guided streaming library used for efficient
 * 3D real-time rendering of highly detailed volumetric scenes.
 *
 * Copyright (C) 2011-2012 INRIA <http://www.inria.fr/>
 *
 * Authors : GigaVoxels Team
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/StaticRes3D.h>
#include <GvStructure/GvVolumeTree.h>
#include <GvRenderer/GvVolumeTreeRendererCUDA.h>
#include <GvRenderer/GvGraphicsInteroperabiltyHandler.h>
#include <GvPerfMon/CUDAPerfMon.h>
#include <GvCore/GvError.h>

// Project
#include "Producer.h"
#include "Shader.h"
#include "ParticleSystem.h"

// GvViewer
#include <GvvApplication.h>
#include <GvvMainWindow.h>

// Cuda SDK
#include <hip/hip_vector_types.h>

// System
#include <cstdlib>
#include <ctime>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

// GigaVoxels
using namespace GvRenderer;

// GigaVoxels viewer
using namespace GvViewerCore;

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

// Defines the size allowed for each type of pool
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		// 8 Mo
#define BRICKPOOL_MEMSIZE	( 256U * 1024U * 1024U )	// 256 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	GvvPipelineInterface()
,	_volumeTree( NULL )
,	_volumeTreeCache( NULL )
,	_volumeTreeRenderer( NULL )
,	_producer( NULL )
,	_colorTex( 0 )
,	_depthTex( 0 )
,	_frameBuffer( 0 )
,	_depthBuffer( 0 )
,	_displayOctree( false )
,	_displayPerfmon( 0 )
,	_maxVolTreeDepth( 7 )
{
	// Translation used to position the GigaVoxels data structure
	_translation[ 0 ] = -0.5f;
	_translation[ 1 ] = -0.5f;
	_translation[ 2 ] = -0.5f;

	// Light position
	_lightPosition = make_float3( 1.f, 1.f, 1.f );

	_shaderOpacityCorrection = 512.0f;
	_brickNbPoints = 1;
	_hasBrickDrawOneSlice = true;
	_hasBrickGigaVoxelsRendering = false;
	_hasBrickOpenGLVBORendering = true;
	_particleSystem = NULL;
	_voxelScale = 1.333f;
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	delete _volumeTreeRenderer;
	delete _volumeTreeCache;
	delete _volumeTree;
	delete _producer;

	delete _particleSystem;
}

/******************************************************************************
 * Gets the name of this browsable
 *
 * @return the name of this browsable
 ******************************************************************************/
const char* SampleCore::getName() const
{
	return "VoxelsVsVBOPoints";
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 ******************************************************************************/
void SampleCore::init()
{
	CUDAPM_INIT();

	// Initialize CUDA with OpenGL Interoperability
	if ( ! GvViewerGui::GvvApplication::get().isGPUComputingInitialized() )
	{
		//cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );	// to do : deprecated, use hipSetDevice()
		//GV_CHECK_CUDA_ERROR( "cudaGLSetGLDevice" );
		hipSetDevice( gpuGetMaxGflopsDeviceId() );
		GV_CHECK_CUDA_ERROR( "hipSetDevice" );
		
		GvViewerGui::GvvApplication::get().setGPUComputingInitialized( true );
	}

	// Compute the size of one element in the cache for nodes and bricks
	size_t nodeElemSize = NodeRes::numElements * sizeof( GvStructure::OctreeNode );
	size_t brickElemSize = RealBrickRes::numElements * GvCore::DataTotalChannelSize< DataType >::value;

	// Compute how many we can fit into the given memory size
	size_t nodePoolNumElems = NODEPOOL_MEMSIZE / nodeElemSize;
	size_t brickPoolNumElems = BRICKPOOL_MEMSIZE / brickElemSize;

	// Compute the resolution of the pools
	uint3 nodePoolRes = make_uint3( static_cast< uint >( floorf( powf( static_cast< float >( nodePoolNumElems ), 1.0f / 3.0f ) ) ) ) * NodeRes::get();
	uint3 brickPoolRes = make_uint3( static_cast< uint >( floorf( powf( static_cast< float >( brickPoolNumElems ), 1.0f / 3.0f ) ) ) ) * RealBrickRes::get();
	
	std::cout << "\nnodePoolRes: " << nodePoolRes << std::endl;
	std::cout << "brickPoolRes: " << brickPoolRes << std::endl;

	// Producer initialization
	_producer = new ProducerType();

	// Data structure initialization
	_volumeTree = new VolumeTreeType( nodePoolRes, brickPoolRes, 0 );
	_volumeTree->setMaxDepth( _maxVolTreeDepth );

	// Cache initialization
	_volumeTreeCache = new VolumeTreeCacheType( _volumeTree, _producer, nodePoolRes, brickPoolRes );

	// Renderer initialization
	_volumeTreeRenderer = new VolumeTreeRendererType( _volumeTree, _volumeTreeCache, _producer );

	// TEST
	srand( time( NULL ) );
	initializeBrick();
	_producer->setBrickDrawOneSlice( hasBrickDrawOneSlice() );
	_producer->setBrickPresenceFlags( _presenceFlags );
	_particleSystem = new ParticleSystem();
	_particleSystem->initialize();
	_particleSystem->setBrickDrawOneSlice( hasBrickDrawOneSlice() );
	_particleSystem->setBrickPresenceFlags( _presenceFlags );
	_particleSystem->load();
}

/******************************************************************************
 * Draw function called of frame
 ******************************************************************************/
void SampleCore::draw()
{
	if ( _hasBrickGigaVoxelsRendering )
	{
		CUDAPM_START_FRAME;
		CUDAPM_START_EVENT( frame );
		CUDAPM_START_EVENT( app_init_frame );

		glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );

		glMatrixMode( GL_MODELVIEW );

		if ( _displayOctree )
		{
			glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );

			// Display the GigaVoxels N3-tree space partitioning structure
			glEnable( GL_DEPTH_TEST );
			glPushMatrix();
			// Translation used to position the GigaVoxels data structure
			glTranslatef( _translation[ 0 ], _translation[ 1 ], _translation[ 2 ] );
			_volumeTree->displayDebugOctree();
			glPopMatrix();
			glDisable( GL_DEPTH_TEST );

			// Clear the depth PBO (pixel buffer object) by reading from the previously cleared FBO (frame buffer object)
			glBindBuffer( GL_PIXEL_PACK_BUFFER, _depthBuffer );
			glReadPixels( 0, 0, _width, _height, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0 );
			glBindBuffer( GL_PIXEL_PACK_BUFFER, 0 );
			GV_CHECK_GL_ERROR();
		}
		else
		{
			glClear( GL_COLOR_BUFFER_BIT );
		}

		glBindFramebuffer( GL_FRAMEBUFFER, 0 );

		// extract view transformations
		float4x4 viewMatrix;
		float4x4 projectionMatrix;
		glGetFloatv( GL_MODELVIEW_MATRIX, viewMatrix._array );
		glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );

		// extract viewport
		GLint params[4];
		glGetIntegerv( GL_VIEWPORT, params );
		int4 viewport = make_int4(params[0], params[1], params[2], params[3]);

		// render the scene into textures
		CUDAPM_STOP_EVENT( app_init_frame );

		// Build the world transformation matrix
		float4x4 modelMatrix;
		glPushMatrix();
		glLoadIdentity();
		// Translation used to position the GigaVoxels data structure
			glTranslatef( _translation[ 0 ], _translation[ 1 ], _translation[ 2 ] );
		glGetFloatv( GL_MODELVIEW_MATRIX, modelMatrix._array );
		glPopMatrix();

		// Render
		_volumeTreeRenderer->render( modelMatrix, viewMatrix, projectionMatrix, viewport );

		// Render the result to the screen
		glMatrixMode( GL_MODELVIEW );
		glPushMatrix();
		glLoadIdentity();

		glMatrixMode( GL_PROJECTION );
		glPushMatrix();
		glLoadIdentity();

		glDisable( GL_DEPTH_TEST );
		glEnable( GL_TEXTURE_RECTANGLE_EXT );
		glActiveTexture( GL_TEXTURE0 );
		glBindTexture( GL_TEXTURE_RECTANGLE_EXT, _colorTex );

		// Draw a full screen quad
		GLint sMin = 0;
		GLint tMin = 0;
		GLint sMax = _width;
		GLint tMax = _height;
		glBegin( GL_QUADS );
		glColor3f( 1.0f, 1.0f, 1.0f );
		glTexCoord2i( sMin, tMin ); glVertex2i( -1, -1 );
		glTexCoord2i( sMax, tMin ); glVertex2i(  1, -1 );
		glTexCoord2i( sMax, tMax ); glVertex2i(  1,  1 );
		glTexCoord2i( sMin, tMax ); glVertex2i( -1,  1 );
		glEnd();

		glActiveTexture( GL_TEXTURE0 );
		glBindTexture( GL_TEXTURE_RECTANGLE_EXT, 0 );
		glDisable( GL_TEXTURE_RECTANGLE_EXT );
	
		glPopMatrix();
		glMatrixMode( GL_MODELVIEW );
		glPopMatrix();

		// TEST - optimization due to early unmap() graphics resource from GigaVoxels
		//_volumeTreeRenderer->doPostRender();
	
		// Update GigaVoxels info
		_volumeTreeRenderer->nextFrame();

		CUDAPM_STOP_EVENT( frame );
		CUDAPM_STOP_FRAME;

		// Display the GigaVoxels performance monitor (if it has been activated during GigaVoxels compilation)
		if ( _displayPerfmon )
		{
			GvPerfMon::CUDAPerfMon::getApplicationPerfMon().displayFrameGL( _displayPerfmon - 1 );
		}
	}
	else // _hasBrickOpenGLVBORendering
	{
		_particleSystem->render();
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param width the new width
 * @param height the new height
 ******************************************************************************/
void SampleCore::resize( int width, int height )
{
	_width = width;
	_height = height;

	// Reset default active frame region for rendering
	_volumeTreeRenderer->setProjectedBBox( make_uint4( 0, 0, _width, _height ) );

	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( _width, _height ) );

	// Create frame-dependent objects
	
	// Disconnect all registered graphics resources
	_volumeTreeRenderer->resetGraphicsResources();
	
	// ...
	if (_depthBuffer)
	{
		glDeleteBuffers(1, &_depthBuffer);
	}

	if (_colorTex)
	{
		glDeleteTextures(1, &_colorTex);
	}
	if (_depthTex)
	{
		glDeleteTextures(1, &_depthTex);
	}

	if (_frameBuffer)
	{
		glDeleteFramebuffers(1, &_frameBuffer);
	}

	glGenTextures(1, &_colorTex);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, _colorTex);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	GV_CHECK_GL_ERROR();

	glGenBuffers(1, &_depthBuffer);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, _depthBuffer);
	glBufferData(GL_PIXEL_PACK_BUFFER, width * height * sizeof(GLuint), NULL, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);
	GV_CHECK_GL_ERROR();

	glGenTextures(1, &_depthTex);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, _depthTex);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_RECTANGLE_EXT, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexImage2D(GL_TEXTURE_RECTANGLE_EXT, 0, GL_DEPTH24_STENCIL8_EXT, width, height, 0, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, NULL);
	glBindTexture(GL_TEXTURE_RECTANGLE_EXT, 0);
	GV_CHECK_GL_ERROR();

	glGenFramebuffers( 1, &_frameBuffer );
	glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_RECTANGLE_EXT, _colorTex, 0 );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_RECTANGLE_EXT, _depthTex, 0 );
	glFramebufferTexture2D( GL_FRAMEBUFFER, GL_STENCIL_ATTACHMENT, GL_TEXTURE_RECTANGLE_EXT, _depthTex, 0 );
	glBindFramebuffer( GL_FRAMEBUFFER, 0 );
	GV_CHECK_GL_ERROR();

	// Create CUDA resources from OpenGL objects
	if ( _displayOctree )
	{
		_volumeTreeRenderer->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		_volumeTreeRenderer->connect( GvGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		_volumeTreeRenderer->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * Clear the GigaVoxels cache
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->clear();
}

/******************************************************************************
 * Toggle the display of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	_displayOctree = !_displayOctree;

	// Disconnect all registered graphics resources
	_volumeTreeRenderer->resetGraphicsResources();

	if ( _displayOctree )
	{
		_volumeTreeRenderer->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		_volumeTreeRenderer->connect( GvGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		_volumeTreeRenderer->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * Toggle the GigaVoxels dynamic update mode
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	const bool status = _pipeline->hasDynamicUpdate();
	_pipeline->setDynamicUpdate( ! status );
}

/******************************************************************************
 * Toggle the display of the performance monitor utility if
 * GigaVoxels has been compiled with the Performance Monitor option
 *
 * @param mode The performance monitor mode (1 for CPU, 2 for DEVICE)
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if ( _displayPerfmon )
	{
		_displayPerfmon = 0;
	}
	else
	{
		_displayPerfmon = mode;
	}
}

/******************************************************************************
 * Increment the max resolution of the data structure
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth < 32 )
	{
		_maxVolTreeDepth++;
	}

	_volumeTree->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Decrement the max resolution of the data structure
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth > 0 )
	{
		_maxVolTreeDepth--;
	}

	_volumeTree->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Get the node tile resolution of the data structure.
 *
 * @param pX the X node tile resolution
 * @param pY the Y node tile resolution
 * @param pZ the Z node tile resolution
 ******************************************************************************/
void SampleCore::getDataStructureNodeTileResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& nodeTileResolution = _volumeTree->getNodeTileResolution().get();

	pX = nodeTileResolution.x;
	pY = nodeTileResolution.y;
	pZ = nodeTileResolution.z;
}

/******************************************************************************
 * Get the brick resolution of the data structure (voxels).
 *
 * @param pX the X brick resolution
 * @param pY the Y brick resolution
 * @param pZ the Z brick resolution
 ******************************************************************************/
void SampleCore::getDataStructureBrickResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& brickResolution = _volumeTree->getBrickResolution().get();

	pX = brickResolution.x;
	pY = brickResolution.y;
	pZ = brickResolution.z;
}

/******************************************************************************
 * Get the max depth.
 *
 * @return the max depth
 ******************************************************************************/
unsigned int SampleCore::getRendererMaxDepth() const
{
	return _volumeTree->getMaxDepth();
}

/******************************************************************************
 * Set the max depth.
 *
 * @param pValue the max depth
 ******************************************************************************/
void SampleCore::setRendererMaxDepth( unsigned int pValue )
{
	_volumeTree->setMaxDepth( pValue );
}

/******************************************************************************
 * Get the max number of requests of node subdivisions.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbNodeSubdivisions() const
{
	return _volumeTreeCache->getMaxNbNodeSubdivisions();
}

/******************************************************************************
 * Set the max number of requests of node subdivisions.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbNodeSubdivisions( unsigned int pValue )
{
	_volumeTreeCache->setMaxNbNodeSubdivisions( pValue );
}

/******************************************************************************
 * Get the max number of requests of brick of voxel loads.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbBrickLoads() const
{
	return _volumeTreeCache->getMaxNbBrickLoads();
}

/******************************************************************************
 * Set the max number of requests of brick of voxel loads.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbBrickLoads( unsigned int pValue )
{
	_volumeTreeCache->setMaxNbBrickLoads( pValue );
}

/******************************************************************************
 * Set the request strategy indicating if, during data structure traversal,
 * priority of requests is set on brick loads or on node subdivisions first.
 *
 * @param pFlag the flag indicating the request strategy
 ******************************************************************************/
void SampleCore::setRendererPriorityOnBricks( bool pFlag )
{
	_volumeTreeRenderer->setPriorityOnBricks( pFlag );
}

/******************************************************************************
 * Specify color to clear the color buffer
 *
 * @param pRed red component
 * @param pGreen green component
 * @param pBlue blue component
 * @param pAlpha alpha component
 ******************************************************************************/
void SampleCore::setClearColor( unsigned char pRed, unsigned char pGreen, unsigned char pBlue, unsigned char pAlpha )
{
	_volumeTreeRenderer->setClearColor( make_uchar4( pRed, pGreen, pBlue, pAlpha ) );
}

/******************************************************************************
 * Tell wheter or not the pipeline has a light.
 *
 * @return the flag telling wheter or not the pipeline has a light
 ******************************************************************************/
bool SampleCore::hasLight() const
{
	return true;
}

/******************************************************************************
 * Get the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::getLightPosition( float& pX, float& pY, float& pZ ) const
{
	pX = _lightPosition.x;
	pY = _lightPosition.y;
	pZ = _lightPosition.z;
}

///******************************************************************************
// * Set the light position
// *
// * @param pX the X light position
// * @param pY the Y light position
// * @param pZ the Z light position
// ******************************************************************************/
//void SampleCore::setLightPosition( float pX, float pY, float pZ )
//{
//	// Update DEVICE memory with "light position"
//	//
//	// WARNING
//	// Apply inverse modelisation matrix applied on the GigaVoxels object to set light position correctly.
//	// Here a glTranslatef( -0.5f, -0.5f, -0.5f ) has been used.
//	_lightPosition.x = pX/* - _translation[ 0 ]*/;
//	_lightPosition.y = pY/* - _translation[ 1 ]*/;
//	_lightPosition.z = pZ/* - _translation[ 2 ]*/;
//
//	// Update device memory
//	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &_lightPosition, sizeof( _lightPosition ), 0, hipMemcpyHostToDevice ) );
//}

/******************************************************************************
 * Set the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{
	// Update DEVICE memory with "light position"
	float3 lightPosition = make_float3( pX, pY, pZ );
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &lightPosition, sizeof( lightPosition ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * Get the translation used to position the GigaVoxels data structure
 *
 * @param pX the x componenet of the translation
 * @param pX the y componenet of the translation
 * @param pX the z componenet of the translation
 ******************************************************************************/
void SampleCore::getTranslation( float& pX, float& pY, float& pZ ) const
{
	pX = _translation[ 0 ];
	pY = _translation[ 1 ];
	pZ = _translation[ 2 ];
}

/******************************************************************************
 * ...
 ******************************************************************************/
float SampleCore::getShaderOpacityCorrection() const
{
	return _shaderOpacityCorrection;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setShaderOpacityCorrection( float pValue )
{
	_shaderOpacityCorrection = pValue;
}

/******************************************************************************
 * ...
 ******************************************************************************/
unsigned int SampleCore::getBrickNbPoints() const
{
	return _brickNbPoints;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setBrickNbPoints( unsigned int pValue )
{
	_brickNbPoints = pValue;

	// Reset brick's probability of presence
	initializeBrick();

	// Update producer
	if ( hasBrickGigaVoxelsRendering() )
	{
		_producer->setBrickPresenceFlags( _presenceFlags );
		_producer->setBrickDrawOneSlice( hasBrickDrawOneSlice() );
		clearCache();
		//_producer->clearCache();
	}
	
	// Update particle system
	if ( hasBrickOpenGLVBORendering() )
	{
		_particleSystem->setBrickNbPoints( pValue );
		_particleSystem->setBrickDrawOneSlice( hasBrickDrawOneSlice() );
		_particleSystem->setBrickPresenceFlags( _presenceFlags );
		_particleSystem->load();
	}
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasBrickDrawOneSlice() const
{
	return _hasBrickDrawOneSlice;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setBrickDrawOneSlice( bool pFlag )
{
	_hasBrickDrawOneSlice = pFlag;

	// Update producer
	if ( hasBrickGigaVoxelsRendering() )
	{
		_producer->setBrickDrawOneSlice( pFlag );
		_producer->setBrickPresenceFlags( _presenceFlags );
		clearCache();
		//_producer->clearCache();
	}

	// Update particle system
	if ( hasBrickOpenGLVBORendering() )
	{
		_particleSystem->setBrickNbPoints( getBrickNbPoints() );
		_particleSystem->setBrickDrawOneSlice( pFlag );
		_particleSystem->setBrickPresenceFlags( _presenceFlags );
		_particleSystem->load();
	}
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasBrickGigaVoxelsRendering() const
{
	return _hasBrickGigaVoxelsRendering;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setBrickGigaVoxelsRendering( bool pFlag )
{
	_hasBrickGigaVoxelsRendering = pFlag;

	// Update producer
	if ( hasBrickGigaVoxelsRendering() )
	{
		_producer->setBrickPresenceFlags( _presenceFlags );
		_producer->setBrickDrawOneSlice( hasBrickDrawOneSlice() );
		clearCache();
		//_producer->clearCache();
	}
	
	//// Update particle system
	//if ( hasBrickOpenGLVBORendering() )
	//{
	//	_particleSystem->setBrickNbPoints( pValue );
	//	_particleSystem->setBrickDrawOneSlice( hasBrickDrawOneSlice() );
	//	_particleSystem->setBrickPresenceFlags( _presenceFlags );
	//	_particleSystem->load();
	//}
}

/******************************************************************************
 * ...
 ******************************************************************************/
bool SampleCore::hasBrickOpenGLVBORendering() const
{
	return _hasBrickOpenGLVBORendering;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setBrickOpenGLVBORendering( bool pFlag )
{
	_hasBrickOpenGLVBORendering = pFlag;
	
	//// Update producer
	//if ( hasBrickGigaVoxelsRendering() )
	//{
	//	_producer->setBrickPresenceFlags( _presenceFlags );
	//	_producer->setBrickDrawOneSlice( hasBrickDrawOneSlice() );
	//	clearCache();
	//	//_producer->clearCache();
	//}
	
	// Update particle system
	if ( hasBrickOpenGLVBORendering() )
	{
		_particleSystem->setBrickNbPoints( getBrickNbPoints() );
		_particleSystem->setBrickDrawOneSlice( hasBrickDrawOneSlice() );
		_particleSystem->setBrickPresenceFlags( _presenceFlags );
		_particleSystem->load();
	}
}

/******************************************************************************
 * ...
 ******************************************************************************/
float SampleCore::getBrickPointSize() const
{
	return _particleSystem->getBrickPointSize();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setBrickPointSize( float pValue )
{
	_particleSystem->setBrickPointSize( pValue);
}

/******************************************************************************
 * ...
 ******************************************************************************/
float SampleCore::getVoxelScale() const
{
	return _voxelScale;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::setVoxelScale( float pValue )
{
	_voxelScale = pValue;

	// Update DEVICE memory with "voxel scale"
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cVoxelScale), &_voxelScale, sizeof( _voxelScale ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::initializeBrick()
{
	// -- Presence flags of points inside a brick --

	const unsigned int brickResolution = BrickRes::x;

	// RESET
	for ( unsigned int z = 0; z < brickResolution; z++ )
	{
		for ( unsigned int y = 0; y < brickResolution; y++ )
		{
			for ( unsigned int x = 0; x < brickResolution; x++ )
			{
				_presenceFlags[ x ][ y ][ z ] = 0;
			}
		}
	}

	// Generate points distribution
	int nbPoints = _brickNbPoints;
	float value = 0.0f;
	while ( nbPoints > 0 )
	{
		for ( unsigned int z = 0; z < brickResolution; z++ )
		{
			for ( unsigned int y = 0; y < brickResolution; y++ )
			{
				for ( unsigned int x = 0; x < brickResolution; x++ )
				{
					if ( nbPoints > 0 )
					{
						if ( _presenceFlags[ x ][ y ][ z ] != 1 )
						{
							value = static_cast< float >( rand() ) / static_cast< float >( RAND_MAX );

							if ( value >= 0.97f )
							{
								_presenceFlags[ x ][ y ][ z ] = 1;

								nbPoints--;

								if ( nbPoints < 1 )
								{
									break;
								}
							}
						}
					}
				}
			}
		}
	}
}
