#include "hip/hip_runtime.h"
/*
 * GigaVoxels is a ray-guided streaming library used for efficient
 * 3D real-time rendering of highly detailed volumetric scenes.
 *
 * Copyright (C) 2011-2013 INRIA <http://www.inria.fr/>
 *
 * Authors : GigaVoxels Team
 *
 * GigaVoxels is distributed under a dual-license scheme.
 * You can obtain a specific license from Inria at gigavoxels-licensing@inria.fr.
 * Otherwise the default license is the GPL version 3.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/StaticRes3D.h>
#include <GvStructure/GvVolumeTree.h>
#include <GvStructure/GvDataProductionManager.h>
#include <GvUtils/GvSimplePipeline.h>
#include <GvUtils/GvSimpleHostProducer.h>
#include <GvUtils/GvDataLoader.h>
#include <GvUtils/GvSimpleHostShader.h>
#include <GvUtils/GvCommonGraphicsPass.h>
#include <GvCore/GvError.h>
#include <GvPerfMon/GvPerformanceMonitor.h>
#include <GvStructure/GvNode.h>

// Project
#include "Producer.h"
//#include "VolumeProducerBricks.h"
#include "VolumeTreeRendererGLSL.h"
//#include "ProducerTorusKernel.h"

// Qt
#include <QCoreApplication>
#include <QString>
#include <QDir>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

/**
 * Defines the size allowed for each type of pool
 */
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		// 8 Mo
#define BRICKPOOL_MEMSIZE	( 128U * 1024U * 1024U )	// 128 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	_pipeline( NULL )
,	_renderer( NULL )
,	mDisplayOctree( false )
,	mDisplayPerfmon( 0 )
,	mMaxVolTreeDepth( 16 )
{
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	delete _pipeline;
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 ******************************************************************************/
void SampleCore::init()
{
	CUDAPM_INIT();

	// Initialize CUDA with OpenGL Interoperability
	//cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );	// to do : deprecated, use hipSetDevice()
	//GV_CHECK_CUDA_ERROR( "cudaGLSetGLDevice" );
	hipSetDevice( gpuGetMaxGflopsDeviceId() );
	GV_CHECK_CUDA_ERROR( "hipSetDevice" );

	// Compute the size of one element in the cache for nodes and bricks
	size_t nodeElemSize = NodeRes::numElements * sizeof( GvStructure::GvNode );
	//size_t brickElemSize = RealBrickRes::numElements * GvCore::DataTotalChannelSize< DataType >::value;

	// Compute how many we can fit into the given memory size
	size_t nodePoolNumElems = NODEPOOL_MEMSIZE / nodeElemSize;
	//size_t brickPoolNumElems = BRICKPOOL_MEMSIZE / brickElemSize;

	// Compute the resolution of the pools
	uint3 nodePoolRes = make_uint3( (uint)floorf( powf( (float)nodePoolNumElems, 1.0f / 3.0f ) ) ) * NodeRes::get();
	//uint3 brickPoolRes = make_uint3( (uint)floorf( powf( (float)brickPoolNumElems, 1.0f / 3.0f ) ) ) * RealBrickRes::get();

	//std::cout << "nodePoolRes: " << nodePoolRes << std::endl;
	//std::cout << "brickPoolRes: " << brickPoolRes << std::endl;

	// Pipeline creation
	_pipeline = new PipelineType();

	// Producer creation
	ProducerType* producer = new ProducerType( 64 * 1024 * 1024, nodePoolRes.x * nodePoolRes.y * nodePoolRes.z );
	QString dataRepository = QCoreApplication::applicationDirPath() + QDir::separator() + QString( "Data" );
	QString filename = dataRepository + QDir::separator() + QString( "Voxels" ) + QDir::separator() + QString( "xyzrgb_dragon512_BR8_B1" ) + QDir::separator() + QString( "xyzrgb_dragon.xml" );
	GvUtils::GvDataLoader< DataType >* dataLoader = new GvUtils::GvDataLoader< DataType >( filename.toStdString(), BrickRes::get(), BrickBorderSize, true );
	producer->attachProducer( dataLoader );

	//ProducerType* producer = new ProducerType();

	// Shader creation
	ShaderType* shader = new ShaderType();

	// Pipeline initialization
	const bool useGraphicsLibraryInteroperability = true;
	_pipeline->initialize( NODEPOOL_MEMSIZE, BRICKPOOL_MEMSIZE, producer, shader, useGraphicsLibraryInteroperability );

	// Renderer initialization
	_renderer = new RendererType( _pipeline->editDataStructure(), _pipeline->editCache() );
	assert( _renderer != NULL );
	_pipeline->addRenderer( _renderer );

	// Pipeline configuration
	_pipeline->editDataStructure()->setMaxDepth( mMaxVolTreeDepth );
}

/******************************************************************************
 * Draw function called of frame
 ******************************************************************************/
void SampleCore::draw()
{
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT( frame );
	CUDAPM_START_EVENT( app_init_frame );

	//glClearColor( 0.0f, 0.1f, 0.3f, 0.0f );
	//glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );

	glEnable( GL_DEPTH_TEST );

	glMatrixMode( GL_MODELVIEW);

	// Display the data structure (space partitioning)
	if ( mDisplayOctree )
	{
		glPushMatrix();
		glTranslatef( -0.5f, -0.5f, -0.5f );
		_pipeline->editDataStructure()->render();
		glPopMatrix();
	}

	// extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	// FIXME
	glPushMatrix();
	glTranslatef( -0.5f, -0.5f, -0.5f );
	// FIXME
	glGetFloatv( GL_MODELVIEW_MATRIX, viewMatrix._array );
	glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );
	// FIXME
	glPopMatrix();

	// build and extract tree transformations
	float4x4 modelMatrix;

	glPushMatrix();
	glLoadIdentity();
	//glTranslatef(-0.5f, -0.5f, -0.5f);
	glGetFloatv( GL_MODELVIEW_MATRIX, modelMatrix._array );
	glPopMatrix();

	// extract viewport
	GLint params[4];
	glGetIntegerv( GL_VIEWPORT, params );
	int4 viewport = make_int4( params[0], params[1], params[2], params[3] );

	CUDAPM_STOP_EVENT( app_init_frame );

	// Render the scene into textures
	_pipeline->execute( modelMatrix, viewMatrix, projectionMatrix, viewport );

	/*_pipeline->editRenderer()*/_renderer->nextFrame();

	CUDAPM_STOP_EVENT( frame );
	CUDAPM_STOP_FRAME;

	// Display the performance monitor
	if ( mDisplayPerfmon )
	{
		GvPerfMon::CUDAPerfMon::get().displayFrameGL( mDisplayPerfmon - 1 );
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param width the new width
 * @param height the new height
 ******************************************************************************/
void SampleCore::resize( int width, int height )
{
	mWidth = width;
	mHeight = height;

	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( mWidth, mHeight ) );

	/*uchar *timersMask = GvPerfMon::CUDAPerfMon::get().getKernelTimerMask();
	hipMemset(timersMask, 255, mWidth * mHeight);*/
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->clear();
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	mDisplayOctree = !mDisplayOctree;
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	const bool status = _pipeline->hasDynamicUpdate();
	_pipeline->setDynamicUpdate( ! status );
}

/******************************************************************************
 * ...
 *
 * @param mode ...
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if ( mDisplayPerfmon )
	{
		mDisplayPerfmon = 0;
	}
	else
	{
		mDisplayPerfmon = mode;
	}
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if ( mMaxVolTreeDepth < 32 )
	{
		mMaxVolTreeDepth++;
	}

	_pipeline->editDataStructure()->setMaxDepth( mMaxVolTreeDepth );
}

/******************************************************************************
 * ...
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if ( mMaxVolTreeDepth > 0 )
	{
		mMaxVolTreeDepth--;
	}

	_pipeline->editDataStructure()->setMaxDepth( mMaxVolTreeDepth );
}

/******************************************************************************
 * Set the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{
	// Update DEVICE memory with "light position"
	//
	float3 lightPos = make_float3( pX, pY, pZ );
	//GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &lightPos, sizeof( lightPos ), 0, hipMemcpyHostToDevice ) );
}
