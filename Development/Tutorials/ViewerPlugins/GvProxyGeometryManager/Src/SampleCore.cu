#include "hip/hip_runtime.h"
/*
 * GigaVoxels is a ray-guided streaming library used for efficient
 * 3D real-time rendering of highly detailed volumetric scenes.
 *
 * Copyright (C) 2011-2013 INRIA <http://www.inria.fr/>
 *
 * Authors : GigaVoxels Team
 *
 * GigaVoxels is distributed under a dual-license scheme.
 * You can obtain a specific license from Inria at gigavoxels-licensing@inria.fr.
 * Otherwise the default license is the GPL version 3.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/** 
 * @version 1.0
 */

#include "SampleCore.h"

/******************************************************************************
 ******************************* INCLUDE SECTION ******************************
 ******************************************************************************/

// GigaVoxels
#include <GvCore/StaticRes3D.h>
#include <GvStructure/GvVolumeTree.h>
#include <GvStructure/GvDataProductionManager.h>
#include <GvUtils/GvSimplePipeline.h>
#include <GvUtils/GvSimpleHostProducer.h>
#include <GvUtils/GvSimpleHostShader.h>
#include <GvUtils/GvCommonGraphicsPass.h>
#include <GvCore/GvError.h>
#include <GvPerfMon/GvPerformanceMonitor.h>

// Project
#include "ProducerKernel.h"
#include "ShaderKernel.h"
#include "RendererCUDA.h"
#include "ProxyGeometry.h"
#include "Mesh.h"

// GvViewer
#include <GvvApplication.h>
#include <GvvMainWindow.h>

// Qt
#include <QCoreApplication>
#include <QString>
#include <QDir>

/******************************************************************************
 ****************************** NAMESPACE SECTION *****************************
 ******************************************************************************/

// GigaVoxels
using namespace GvRendering;
using namespace GvUtils;
using namespace GsGraphics;

// GigaVoxels viewer
using namespace GvViewerCore;

// STL
using namespace std;

/******************************************************************************
 ************************* DEFINE AND CONSTANT SECTION ************************
 ******************************************************************************/

// Defines the size allowed for each type of pool
#define NODEPOOL_MEMSIZE	( 8U * 1024U * 1024U )		// 8 Mo
#define BRICKPOOL_MEMSIZE	( 256U * 1024U * 1024U )	// 256 Mo

/******************************************************************************
 ***************************** TYPE DEFINITION ********************************
 ******************************************************************************/

/******************************************************************************
 ***************************** METHOD DEFINITION ******************************
 ******************************************************************************/

/******************************************************************************
 * Constructor
 ******************************************************************************/
SampleCore::SampleCore()
:	_pipeline( NULL )
,	_graphicsEnvironment( NULL )
,	_displayOctree( false )
,	_displayPerfmon( 0 )
,	_maxVolTreeDepth( 5 )
,	_depthBuffer( 0 )
,	_colorTex( 0 )
,	_depthTex( 0 )
,	_frameBuffer( 0 )
,	_width( 0 )
,	_height( 0 )
,	_proxyGeometry( NULL )
{
	// Translation used to position the GigaVoxels data structure
	_translation[ 0 ] = -0.5f;
	_translation[ 1 ] = -0.5f;
	_translation[ 2 ] = -0.5f;

	// Light position
	_lightPosition = make_float3( 1.f, 1.f, 1.f );
}

/******************************************************************************
 * Destructor
 ******************************************************************************/
SampleCore::~SampleCore()
{
	delete _pipeline;
	delete _graphicsEnvironment;

	delete _proxyGeometry;
}

/******************************************************************************
 * Gets the name of this browsable
 *
 * @return the name of this browsable
 ******************************************************************************/
const char* SampleCore::getName() const
{
	return "Proxy Geometry Manager";
}

/******************************************************************************
 * Initialize the GigaVoxels pipeline
 ******************************************************************************/
void SampleCore::init()
{
	CUDAPM_INIT();

	// Initialize CUDA with OpenGL Interoperability
	if ( ! GvViewerGui::GvvApplication::get().isGPUComputingInitialized() )
	{
		//cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );	// to do : deprecated, use hipSetDevice()
		//GV_CHECK_CUDA_ERROR( "cudaGLSetGLDevice" );
		hipSetDevice( gpuGetMaxGflopsDeviceId() );
		GV_CHECK_CUDA_ERROR( "hipSetDevice" );
		
		GvViewerGui::GvvApplication::get().setGPUComputingInitialized( true );
	}

	// Pipeline creation
	_pipeline = new PipelineType();
	ProducerType* producer = new ProducerType();
	ShaderType* shader = new ShaderType();

	// Pipeline initialization
	_pipeline->initialize( NODEPOOL_MEMSIZE, BRICKPOOL_MEMSIZE, producer, shader );

	// Pipeline configuration
	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );

	// Configure the Cache Management System
	_pipeline->editCache()->setMaxNbNodeSubdivisions( 500 );
	_pipeline->editCache()->setMaxNbBrickLoads( 300 );
	_pipeline->editCache()->editNodesCacheManager()->setPolicy( DataProductionManagerType::NodesCacheManager::eAllPolicies );
	_pipeline->editCache()->editBricksCacheManager()->setPolicy( DataProductionManagerType::BricksCacheManager::eAllPolicies );
	
	// Graphics environment creation
	_graphicsEnvironment = new GvCommonGraphicsPass();

	// Initialize proxy geometry
	//
	// - find a way to modify internal buffer size
	_proxyGeometry = new ProxyGeometry();
	const QString dataRepository = QCoreApplication::applicationDirPath() + QDir::separator() + QString( "Data" );
	const QString meshRepository = dataRepository + QDir::separator() + QString( "3DModels" ) + QDir::separator() + QString( "stanford_bunny" );
	const QString meshFilename = meshRepository + QDir::separator() + QString( "bunny.obj" );
	_proxyGeometry->set3DModelFilename( meshFilename.toStdString() );
	_proxyGeometry->initialize();
	// Restore previous proxy geometry state
	_proxyGeometry->setScreenBasedCriteria( true );
	_proxyGeometry->setScreenBasedCriteriaCoefficient( 45.f );
	_proxyGeometry->setMaterialAlphaCorrectionCoefficient( 512.f );
	// Register proxy geometry
	_pipeline->editRenderer()->setProxyGeometry( _proxyGeometry );

	// Noise parameters
	setNoiseFirstFrequency( 0.1f );
	setNoiseStrength( 1.f );
}

/******************************************************************************
 * Draw function called of frame
 ******************************************************************************/
void SampleCore::draw()
{
	CUDAPM_START_FRAME;
	CUDAPM_START_EVENT( frame );
	CUDAPM_START_EVENT( app_init_frame );

	glMatrixMode( GL_MODELVIEW );
	
	glBindFramebuffer( GL_FRAMEBUFFER, _frameBuffer );
	if ( _displayOctree )
	{
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );

		// Display the GigaVoxels N3-tree space partitioning structure
		glEnable( GL_DEPTH_TEST );
		glPushMatrix();
		glTranslatef( -0.5f, -0.5f, -0.5f );
		_pipeline->editDataStructure()->render();
		glPopMatrix();
		glDisable( GL_DEPTH_TEST );

		// Clear the depth PBO (pixel buffer object) by reading from the previously cleared FBO (frame buffer object)
		glBindBuffer( GL_PIXEL_PACK_BUFFER, _depthBuffer );
		glReadPixels( 0, 0, _width, _height, GL_DEPTH_STENCIL_EXT, GL_UNSIGNED_INT_24_8_EXT, 0 );
		glBindBuffer( GL_PIXEL_PACK_BUFFER, 0 );
		GV_CHECK_GL_ERROR();
	}
	else
	{
		//glClear( GL_COLOR_BUFFER_BIT );
		glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT | GL_STENCIL_BUFFER_BIT );
	}
	glBindFramebuffer( GL_FRAMEBUFFER, 0 );

	// extract view transformations
	float4x4 viewMatrix;
	float4x4 projectionMatrix;
	glGetFloatv( GL_MODELVIEW_MATRIX, viewMatrix._array );
	glGetFloatv( GL_PROJECTION_MATRIX, projectionMatrix._array );

	// extract viewport
	GLint params[ 4 ];
	glGetIntegerv( GL_VIEWPORT, params );
	int4 viewport = make_int4( params[ 0 ], params[ 1 ], params[ 2 ], params[ 3 ] );

	// render the scene into textures
	CUDAPM_STOP_EVENT( app_init_frame );

	// Generate depth maps from mesh
	// - min depth from closest faces
	// - max depth from farthest faces
	// => we get a "shell" from the mesh
	glEnable( GL_DEPTH_TEST );
	glDisable( GL_CULL_FACE );
	float4x4 proxyGeometryModelViewMatrix;
	glPushMatrix();
	// Add Model transformation to lie between -0.5 and 0.5
	const IMesh* mesh = _proxyGeometry->getMesh();
	const float minX = mesh->_minX;
	const float minY = mesh->_minY;
	const float minZ = mesh->_minZ;
	const float maxX = mesh->_maxX;
	const float maxY = mesh->_maxY;
	const float maxZ = mesh->_maxZ;
	const float uniformScale = 0.99f / std::max( std::max( maxX - minX, maxY - minY ), maxZ - minZ );
	glScalef( uniformScale, uniformScale, uniformScale );
	const float3 translate = make_float3( - ( minX + maxX ) * 0.5f, - ( minY + maxY ) * 0.5f, - ( minZ + maxZ ) * 0.5f );
	glTranslatef( translate.x, translate.y, translate.z );
	glGetFloatv( GL_MODELVIEW_MATRIX, proxyGeometryModelViewMatrix._array );
	// TO DO : add a screen based criteria to stop division => ...
	_proxyGeometry->render( proxyGeometryModelViewMatrix, projectionMatrix, viewport );
	glPopMatrix();

	// Build the world transformation matrix
	float4x4 modelMatrix;
	glPushMatrix();
	glLoadIdentity();
	glTranslatef( -0.5f, -0.5f, -0.5f );
	glGetFloatv( GL_MODELVIEW_MATRIX, modelMatrix._array );
	glPopMatrix();

	// Render
	_pipeline->execute( modelMatrix, viewMatrix, projectionMatrix, viewport );

	// Render the result to the screen
	glMatrixMode( GL_MODELVIEW );
	glPushMatrix();
	glLoadIdentity();

	glMatrixMode( GL_PROJECTION );
	glPushMatrix();
	glLoadIdentity();

	glDisable( GL_DEPTH_TEST );
	glEnable( GL_TEXTURE_RECTANGLE_EXT );
	glActiveTexture( GL_TEXTURE0 );
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, _colorTex );
	
	// Draw a full screen quad
	GLint sMin = 0;
	GLint tMin = 0;
	GLint sMax = _width;
	GLint tMax = _height;
	glBegin( GL_QUADS );
	glColor3f( 1.0f, 1.0f, 1.0f );
	glTexCoord2i( sMin, tMin ); glVertex2i( -1, -1 );
	glTexCoord2i( sMax, tMin ); glVertex2i(  1, -1 );
	glTexCoord2i( sMax, tMax ); glVertex2i(  1,  1 );
	glTexCoord2i( sMin, tMax ); glVertex2i( -1,  1 );
	glEnd();

	glActiveTexture( GL_TEXTURE0 );
	glBindTexture( GL_TEXTURE_RECTANGLE_EXT, 0 );
	glDisable( GL_TEXTURE_RECTANGLE_EXT );
	
	glPopMatrix();
	glMatrixMode( GL_MODELVIEW );
	glPopMatrix();

	// TEST - optimization due to early unmap() graphics resource from GigaVoxels
	//_pipeline->editRenderer()->doPostRender();
	
	// Update GigaVoxels info
	_pipeline->editRenderer()->nextFrame();

	CUDAPM_STOP_EVENT( frame );
	CUDAPM_STOP_FRAME;

	// Display the GigaVoxels performance monitor (if it has been activated during GigaVoxels compilation)
	if ( _displayPerfmon )
	{
		GvPerfMon::CUDAPerfMon::get().displayFrameGL( _displayPerfmon - 1 );
	}
}

/******************************************************************************
 * Resize the frame
 *
 * @param width the new width
 * @param height the new height
 ******************************************************************************/
void SampleCore::resize( int pWidth, int pHeight )
{
	// LOG
	//
	// @todo : check and avoid 0 values, replace by 1 and warn user
	if ( pWidth == 0 )
	{
		// TO DO
		// ...
	}
	if ( pHeight == 0 )
	{
		// TO DO
		// ...
	}

	_width = pWidth;
	_height = pHeight;

	// Reset default active frame region for rendering
	_pipeline->editRenderer()->setProjectedBBox( make_uint4( 0, 0, pWidth, pHeight ) );
	// Re-init Perfmon subsystem
	CUDAPM_RESIZE( make_uint2( pWidth, pHeight ) );

	// Update graphics environment
	_graphicsEnvironment->setBufferSize( pWidth, pHeight );

	// Reset graphics resources
	resetGraphicsresources();
}

/******************************************************************************
 * Reset graphics resources
 ******************************************************************************/
void SampleCore::resetGraphicsresources()
{
	// [ 1 ] - Reset graphics resources

	// Disconnect all registered graphics resources
	_pipeline->editRenderer()->resetGraphicsResources();
	
	// Update graphics environment
	_graphicsEnvironment->reset();
	
	// Update internal variables
	_depthBuffer = _graphicsEnvironment->getDepthBuffer();
	_colorTex = _graphicsEnvironment->getColorTexture();
	//_colorRenderBuffer = _graphicsEnvironment->getColorRenderBuffer();
	_depthTex = _graphicsEnvironment->getDepthTexture();
	_frameBuffer = _graphicsEnvironment->getFrameBuffer();
	
	// [ 2 ] - Connect graphics resources

	// Create CUDA resources from OpenGL objects
	if ( _displayOctree )
	{
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}

	// Reset proxy geometry resources
	_pipeline->editRenderer()->unregisterProxyGeometryGraphicsResources();
	_proxyGeometry->setBufferSize( _width, _height );
	_pipeline->editRenderer()->registerProxyGeometryGraphicsResources();
}

/******************************************************************************
 * Clear the GigaVoxels cache
 ******************************************************************************/
void SampleCore::clearCache()
{
	_pipeline->clear();
}

/******************************************************************************
 * Toggle the display of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::toggleDisplayOctree()
{
	_displayOctree = !_displayOctree;

	// Disconnect all registered graphics resources
	_pipeline->editRenderer()->resetGraphicsResources();

	if ( _displayOctree )
	{
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorReadWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eDepthReadSlot, _depthBuffer );
	}
	else
	{
		_pipeline->editRenderer()->connect( GvGraphicsInteroperabiltyHandler::eColorWriteSlot, _colorTex, GL_TEXTURE_RECTANGLE_EXT );
	}
}

/******************************************************************************
 * Get the appearance of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::getDataStructureAppearance( bool& pShowNodeHasBrickTerminal, bool& pShowNodeHasBrickNotTerminal, bool& pShowNodeIsBrickNotInCache, bool& pShowNodeEmptyOrConstant
											, float& pNodeHasBrickTerminalColorR, float& pNodeHasBrickTerminalColorG, float& pNodeHasBrickTerminalColorB, float& pNodeHasBrickTerminalColorA
											, float& pNodeHasBrickNotTerminalColorR, float& pNodeHasBrickNotTerminalColorG, float& pNodeHasBrickNotTerminalColorB, float& pNodeHasBrickNotTerminalColorA
											, float& pNodeIsBrickNotInCacheColorR, float& pNodeIsBrickNotInCacheColorG, float& pNodeIsBrickNotInCacheColorB, float& pNodeIsBrickNotInCacheColorA
											, float& pNodeEmptyOrConstantColorR, float& pNodeEmptyOrConstantColorG, float& pNodeEmptyOrConstantColorB, float& pNodeEmptyOrConstantColorA ) const
{
	float4 nodeHasBrickTerminalColor;
	float4 nodeHasBrickNotTerminalColor;
	float4 nodeIsBrickNotInCacheColor;
	float4 nodeEmptyOrConstantColor;
										
	_pipeline->getDataStructure()->getDataStructureAppearance( pShowNodeHasBrickTerminal, pShowNodeHasBrickNotTerminal, pShowNodeIsBrickNotInCache, pShowNodeEmptyOrConstant
											, nodeHasBrickTerminalColor, nodeHasBrickNotTerminalColor, nodeIsBrickNotInCacheColor, nodeEmptyOrConstantColor );

	pNodeHasBrickTerminalColorR = nodeHasBrickTerminalColor.x;
	pNodeHasBrickTerminalColorG = nodeHasBrickTerminalColor.y;
	pNodeHasBrickTerminalColorB = nodeHasBrickTerminalColor.z;
	pNodeHasBrickTerminalColorA = nodeHasBrickTerminalColor.w;

	pNodeHasBrickNotTerminalColorR = nodeHasBrickNotTerminalColor.x;
	pNodeHasBrickNotTerminalColorG = nodeHasBrickNotTerminalColor.y;
	pNodeHasBrickNotTerminalColorB = nodeHasBrickNotTerminalColor.z;
	pNodeHasBrickNotTerminalColorA = nodeHasBrickNotTerminalColor.w;

	pNodeIsBrickNotInCacheColorR = nodeIsBrickNotInCacheColor.x;
	pNodeIsBrickNotInCacheColorG = nodeIsBrickNotInCacheColor.y;
	pNodeIsBrickNotInCacheColorB = nodeIsBrickNotInCacheColor.z;
	pNodeIsBrickNotInCacheColorA = nodeIsBrickNotInCacheColor.w;

	pNodeEmptyOrConstantColorR = nodeEmptyOrConstantColor.x;
	pNodeEmptyOrConstantColorG = nodeEmptyOrConstantColor.y;
	pNodeEmptyOrConstantColorB = nodeEmptyOrConstantColor.z;
	pNodeEmptyOrConstantColorA = nodeEmptyOrConstantColor.w;
}

/******************************************************************************
 * Set the appearance of the N-tree (octree) of the data structure
 ******************************************************************************/
void SampleCore::setDataStructureAppearance( bool pShowNodeHasBrickTerminal, bool pShowNodeHasBrickNotTerminal, bool pShowNodeIsBrickNotInCache, bool pShowNodeEmptyOrConstant
											, float pNodeHasBrickTerminalColorR, float pNodeHasBrickTerminalColorG, float pNodeHasBrickTerminalColorB, float pNodeHasBrickTerminalColorA
											, float pNodeHasBrickNotTerminalColorR, float pNodeHasBrickNotTerminalColorG, float pNodeHasBrickNotTerminalColorB, float pNodeHasBrickNotTerminalColorA
											, float pNodeIsBrickNotInCacheColorR, float pNodeIsBrickNotInCacheColorG, float pNodeIsBrickNotInCacheColorB, float pNodeIsBrickNotInCacheColorA
											, float pNodeEmptyOrConstantColorR, float pNodeEmptyOrConstantColorG, float pNodeEmptyOrConstantColorB, float pNodeEmptyOrConstantColorA )
{
	float4 nodeHasBrickTerminalColor = make_float4( pNodeHasBrickTerminalColorR, pNodeHasBrickTerminalColorG, pNodeHasBrickTerminalColorB, pNodeHasBrickTerminalColorA );
	float4 nodeHasBrickNotTerminalColor = make_float4( pNodeHasBrickNotTerminalColorR, pNodeHasBrickNotTerminalColorG, pNodeHasBrickNotTerminalColorB, pNodeHasBrickNotTerminalColorA );
	float4 nodeIsBrickNotInCacheColor = make_float4( pNodeIsBrickNotInCacheColorR, pNodeIsBrickNotInCacheColorG, pNodeIsBrickNotInCacheColorB, pNodeIsBrickNotInCacheColorA );
	float4 nodeEmptyOrConstantColor = make_float4( pNodeEmptyOrConstantColorR, pNodeEmptyOrConstantColorG, pNodeEmptyOrConstantColorB, pNodeEmptyOrConstantColorA );

	_pipeline->editDataStructure()->setDataStructureAppearance( pShowNodeHasBrickTerminal, pShowNodeHasBrickNotTerminal, pShowNodeIsBrickNotInCache, pShowNodeEmptyOrConstant
											, nodeHasBrickTerminalColor, nodeHasBrickNotTerminalColor, nodeIsBrickNotInCacheColor, nodeEmptyOrConstantColor );
}

/******************************************************************************
 * Toggle the GigaVoxels dynamic update mode
 ******************************************************************************/
void SampleCore::toggleDynamicUpdate()
{
	const bool status = _pipeline->hasDynamicUpdate();
	_pipeline->setDynamicUpdate( ! status );
}

/******************************************************************************
 * Toggle the display of the performance monitor utility if
 * GigaVoxels has been compiled with the Performance Monitor option
 *
 * @param mode The performance monitor mode (1 for CPU, 2 for DEVICE)
 ******************************************************************************/
void SampleCore::togglePerfmonDisplay( uint mode )
{
	if ( _displayPerfmon )
	{
		_displayPerfmon = 0;
	}
	else
	{
		_displayPerfmon = mode;
	}
}

/******************************************************************************
 * Increment the max resolution of the data structure
 ******************************************************************************/
void SampleCore::incMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth < 32 )
	{
		_maxVolTreeDepth++;
	}

	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Decrement the max resolution of the data structure
 ******************************************************************************/
void SampleCore::decMaxVolTreeDepth()
{
	if ( _maxVolTreeDepth > 0 )
	{
		_maxVolTreeDepth--;
	}

	_pipeline->editDataStructure()->setMaxDepth( _maxVolTreeDepth );
}

/******************************************************************************
 * Get the node tile resolution of the data structure.
 *
 * @param pX the X node tile resolution
 * @param pY the Y node tile resolution
 * @param pZ the Z node tile resolution
 ******************************************************************************/
void SampleCore::getDataStructureNodeTileResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& nodeTileResolution = _pipeline->editDataStructure()->getNodeTileResolution().get();

	pX = nodeTileResolution.x;
	pY = nodeTileResolution.y;
	pZ = nodeTileResolution.z;
}

/******************************************************************************
 * Get the brick resolution of the data structure (voxels).
 *
 * @param pX the X brick resolution
 * @param pY the Y brick resolution
 * @param pZ the Z brick resolution
 ******************************************************************************/
void SampleCore::getDataStructureBrickResolution( unsigned int& pX, unsigned int& pY, unsigned int& pZ ) const
{
	const uint3& brickResolution = _pipeline->editDataStructure()->getBrickResolution().get();

	pX = brickResolution.x;
	pY = brickResolution.y;
	pZ = brickResolution.z;
}

/******************************************************************************
 * Get the max depth.
 *
 * @return the max depth
 ******************************************************************************/
unsigned int SampleCore::getRendererMaxDepth() const
{
	return _pipeline->editDataStructure()->getMaxDepth();
}

/******************************************************************************
 * Set the max depth.
 *
 * @param pValue the max depth
 ******************************************************************************/
void SampleCore::setRendererMaxDepth( unsigned int pValue )
{
	_pipeline->editDataStructure()->setMaxDepth( pValue );
}

/******************************************************************************
 * Get the max number of requests of node subdivisions.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbNodeSubdivisions() const
{
	return _pipeline->getCache()->getMaxNbNodeSubdivisions();
}

/******************************************************************************
 * Set the max number of requests of node subdivisions.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbNodeSubdivisions( unsigned int pValue )
{
	_pipeline->editCache()->setMaxNbNodeSubdivisions( pValue );
}

/******************************************************************************
 * Get the max number of requests of brick of voxel loads.
 *
 * @return the max number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheMaxNbBrickLoads() const
{
	return _pipeline->getCache()->getMaxNbBrickLoads();
}

/******************************************************************************
 * Set the max number of requests of brick of voxel loads.
 *
 * @param pValue the max number of requests
 ******************************************************************************/
void SampleCore::setCacheMaxNbBrickLoads( unsigned int pValue )
{
	_pipeline->editCache()->setMaxNbBrickLoads( pValue );
}

/******************************************************************************
 * Set the request strategy indicating if, during data structure traversal,
 * priority of requests is set on brick loads or on node subdivisions first.
 *
 * @param pFlag the flag indicating the request strategy
 ******************************************************************************/
void SampleCore::setRendererPriorityOnBricks( bool pFlag )
{
	_pipeline->editRenderer()->setPriorityOnBricks( pFlag );
}

/******************************************************************************
 * Specify color to clear the color buffer
 *
 * @param pRed red component
 * @param pGreen green component
 * @param pBlue blue component
 * @param pAlpha alpha component
 ******************************************************************************/
void SampleCore::setClearColor( unsigned char pRed, unsigned char pGreen, unsigned char pBlue, unsigned char pAlpha )
{
	_pipeline->editRenderer()->setClearColor( make_uchar4( pRed, pGreen, pBlue, pAlpha ) );
}

/******************************************************************************
 * Tell wheter or not the pipeline has a light.
 *
 * @return the flag telling wheter or not the pipeline has a light
 ******************************************************************************/
bool SampleCore::hasLight() const
{
	return false;
}

/******************************************************************************
 * Get the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::getLightPosition( float& pX, float& pY, float& pZ ) const
{
	pX = _lightPosition.x;
	pY = _lightPosition.y;
	pZ = _lightPosition.z;
}

/******************************************************************************
 * Set the light position
 *
 * @param pX the X light position
 * @param pY the Y light position
 * @param pZ the Z light position
 ******************************************************************************/
void SampleCore::setLightPosition( float pX, float pY, float pZ )
{
	// Update DEVICE memory with "light position"
	//
	// WARNING
	// Apply inverse modelisation matrix applied on the GigaVoxels object to set light position correctly.
	// Here a glTranslatef( -0.5f, -0.5f, -0.5f ) has been used.
	_lightPosition.x = pX/* - _translation[ 0 ]*/;
	_lightPosition.y = pY/* - _translation[ 1 ]*/;
	_lightPosition.z = pZ/* - _translation[ 2 ]*/;

	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cLightPosition), &_lightPosition, sizeof( _lightPosition ), 0, hipMemcpyHostToDevice ) );
}

/******************************************************************************
 * Get the translation used to position the GigaVoxels data structure
 *
 * @param pX the x componenet of the translation
 * @param pX the y componenet of the translation
 * @param pX the z componenet of the translation
 ******************************************************************************/
void SampleCore::getTranslation( float& pX, float& pY, float& pZ ) const
{
	pX = _translation[ 0 ];
	pY = _translation[ 1 ];
	pZ = _translation[ 2 ];
}

/******************************************************************************
 * Get the number of requests of node subdivisions the cache has handled.
 *
 * @return the number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheNbNodeSubdivisionRequests() const
{
	return _pipeline->getCache()->getNbNodeSubdivisionRequests();
}

/******************************************************************************
 * Get the number of requests of brick of voxel loads the cache has handled.
 *
 * @return the number of requests
 ******************************************************************************/
unsigned int SampleCore::getCacheNbBrickLoadRequests() const
{
	return _pipeline->getCache()->getNbBrickLoadRequests();
}

/******************************************************************************
 * Get the cache policy
 *
 * @return the cache policy
 ******************************************************************************/
unsigned int SampleCore::getCachePolicy() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getPolicy();
}

/******************************************************************************
 * Set the cache policy
 *
 * @param pValue the cache policy
 ******************************************************************************/
void SampleCore::setCachePolicy( unsigned int pValue )
{
	_pipeline->editCache()->editNodesCacheManager()->setPolicy( static_cast< DataProductionManagerType::NodesCacheManager::ECachePolicy >( pValue ) );
	_pipeline->editCache()->editBricksCacheManager()->setPolicy( static_cast< DataProductionManagerType::BricksCacheManager::ECachePolicy >( pValue ) );
}

/******************************************************************************
 * Get the node cache memory
 *
 * @return the node cache memory
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheMemory() const
{
	return NODEPOOL_MEMSIZE / ( 1024U * 1024U );
}

/******************************************************************************
 * Set the node cache memory
 *
 * @param pValue the node cache memory
 ******************************************************************************/
void SampleCore::setNodeCacheMemory( unsigned int pValue )
{
}

/******************************************************************************
 * Get the brick cache memory
 *
 * @return the brick cache memory
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheMemory() const
{
	return BRICKPOOL_MEMSIZE / ( 1024U * 1024U );
}

/******************************************************************************
 * Set the brick cache memory
 *
 * @param pValue the brick cache memory
 ******************************************************************************/
void SampleCore::setBrickCacheMemory( unsigned int pValue )
{
}

/******************************************************************************
 * Get the node cache capacity
 *
 * @return the node cache capacity
 ******************************************************************************/
unsigned int SampleCore::getNodeCacheCapacity() const
{
	return _pipeline->getCache()->getNodesCacheManager()->getNumElements();
}

/******************************************************************************
 * Set the node cache capacity
 *
 * @param pValue the node cache capacity
 ******************************************************************************/
void SampleCore::setNodeCacheCapacity( unsigned int pValue )
{
}

/******************************************************************************
 * Get the brick cache capacity
 *
 * @return the brick cache capacity
 ******************************************************************************/
unsigned int SampleCore::getBrickCacheCapacity() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getNumElements();
}

/******************************************************************************
 * Set the brick cache capacity
 *
 * @param pValue the brick cache capacity
 ******************************************************************************/
void SampleCore::setBrickCacheCapacity( unsigned int pValue )
{
}

/******************************************************************************
 * Get the number of unused nodes in cache
 *
 * @return the number of unused nodes in cache
 ******************************************************************************/
unsigned int SampleCore::getCacheNbUnusedNodes() const
{
	return _pipeline->getCache()->getNodesCacheManager()->getNbUnusedElements();
}

/******************************************************************************
 * Get the number of unused bricks in cache
 *
 * @return the number of unused bricks in cache
 ******************************************************************************/
unsigned int SampleCore::getCacheNbUnusedBricks() const
{
	return _pipeline->getCache()->getBricksCacheManager()->getNbUnusedElements();
}

/******************************************************************************
 * Tell wheter or not the pipeline has a light.
 *
 * @return the flag telling wheter or not the pipeline has a light
 ******************************************************************************/
bool SampleCore::has3DModel() const
{
	return true;
}

/******************************************************************************
 * Get the 3D model filename to load
 *
 * @return the 3D model filename to load
 ******************************************************************************/
string SampleCore::get3DModelFilename() const
{
	return _proxyGeometry->get3DModelFilename();
}

/******************************************************************************
 * Set the 3D model filename to load
 *
 * @param pFilename the 3D model filename to load
 ******************************************************************************/
void SampleCore::set3DModelFilename( const string& pFilename )
{
	// Store current proxy geometry state
	const bool screenBasedCriteria =_proxyGeometry->getScreenBasedCriteria();
	const float screenBasedCriteriaCoefficient =_proxyGeometry->getScreenBasedCriteriaCoefficient();
	const float materialAlphaCorrectionCoefficient =_proxyGeometry->getMaterialAlphaCorrectionCoefficient();
	
	// ---- Delete the 3D scene if needed ----
	
	if ( _proxyGeometry != NULL )
	{
		delete _proxyGeometry;
		_proxyGeometry = NULL;

		// Clear the GigaVoxels cache
		_pipeline->editCache()->clearCache();
	}

	// Initialize proxy geometry (load the 3D scene)
	//
	// - find a way to modify internal buffer size
	_proxyGeometry = new ProxyGeometry();
	_proxyGeometry->set3DModelFilename( pFilename );
	_proxyGeometry->initialize();
	// Restore previous proxy geometry state
	_proxyGeometry->setScreenBasedCriteria( screenBasedCriteria );
	_proxyGeometry->setScreenBasedCriteriaCoefficient( screenBasedCriteriaCoefficient );
	_proxyGeometry->setMaterialAlphaCorrectionCoefficient( materialAlphaCorrectionCoefficient );
	_pipeline->editRenderer()->setProxyGeometry( _proxyGeometry );
	// Reset proxy geometry resources
	_pipeline->editRenderer()->unregisterProxyGeometryGraphicsResources();
	_proxyGeometry->setBufferSize( _width, _height );
	_pipeline->editRenderer()->registerProxyGeometryGraphicsResources();
	// Noise parameters
	setNoiseFirstFrequency( _noiseFirstFrequency );
	setNoiseStrength( _noiseStrength );
}

/******************************************************************************
 * Tell wheter or not the screen-based criteria is activated
 *
 * @return a flag telling wheter or not the screen-based criteria is activated
 ******************************************************************************/
bool SampleCore::getScreenBasedCriteria() const
{
	assert( _proxyGeometry != NULL );

	return _proxyGeometry->getScreenBasedCriteria();
}

/******************************************************************************
 * Set the flag telling wheter or not the screen-based criteria is activated
 *
 * @param pFlag a flag telling wheter or not the screen-based criteria is activated
 ******************************************************************************/
void SampleCore::setScreenBasedCriteria( bool pFlag )
{
	assert( _proxyGeometry != NULL );

	_proxyGeometry->setScreenBasedCriteria( pFlag );

	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cScreenBasedCriteria), &pFlag, sizeof( pFlag ), 0, hipMemcpyHostToDevice ) );

	// Clear the GigaVoxels cache
	clearCache();
}

/******************************************************************************
 * Get the screen-based criteria coefficient
 *
 * @return the screen-based criteria coefficient
 ******************************************************************************/
float SampleCore::getScreenBasedCriteriaCoefficient() const
{
	assert( _proxyGeometry != NULL );

	return _proxyGeometry->getScreenBasedCriteriaCoefficient();
}

/******************************************************************************
 * Set the screen-based criteria coefficient
 *
 * @param pValue the screen-based criteria coefficient
 ******************************************************************************/
void SampleCore::setScreenBasedCriteriaCoefficient( float pValue )
{
	assert( _proxyGeometry != NULL );

	_proxyGeometry->setScreenBasedCriteriaCoefficient( pValue );

	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cScreenBasedCriteriaCoefficient), &pValue, sizeof( pValue ), 0, hipMemcpyHostToDevice ) );

	// Clear the GigaVoxels cache
	clearCache();
}

/******************************************************************************
 * Get the material alpha correction coefficient
 *
 * @return the material alpha correction coefficient
 ******************************************************************************/
float SampleCore::getMaterialAlphaCorrectionCoefficient() const
{
	assert( _proxyGeometry != NULL );

	return _proxyGeometry->getMaterialAlphaCorrectionCoefficient();
}

/******************************************************************************
 * Set the material alpha correction coefficient
 *
 * @param pValue the material alpha correction coefficient
 ******************************************************************************/
void SampleCore::setMaterialAlphaCorrectionCoefficient( float pValue )
{
	assert( _proxyGeometry != NULL );

	_proxyGeometry->setMaterialAlphaCorrectionCoefficient( pValue );

	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cMaterialAlphaCorrectionCoefficient), &pValue, sizeof( pValue ), 0, hipMemcpyHostToDevice ) );

	// No need to clear the cache
	// - this variable is evaluated in the shader, not in the producer.
}

/******************************************************************************
 * Get the noise first frequency
 *
 * @return the noise first frequency
 ******************************************************************************/
float SampleCore::getNoiseFirstFrequency() const
{
	return _noiseFirstFrequency;
}

/******************************************************************************
 * Set the noise first frequency
 *
 * @param pValue the noise first frequency
 ******************************************************************************/
void SampleCore::setNoiseFirstFrequency( float pValue )
{
	_noiseFirstFrequency = pValue;
	
	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNoiseFirstFrequency), &_noiseFirstFrequency, sizeof( _noiseFirstFrequency ), 0, hipMemcpyHostToDevice ) );

	// Clear cache
	clearCache();
}

/******************************************************************************
 * Get the noise strength
 *
 * @return the noise strength
 ******************************************************************************/
float SampleCore::getNoiseStrength() const
{
	return _noiseStrength;
}

/******************************************************************************
 * Set the noise strength
 *
 * @param pValue the noise strength
 ******************************************************************************/
void SampleCore::setNoiseStrength( float pValue )
{
	_noiseStrength = pValue;
	
	// Update device memory
	GV_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( cNoiseStrength), &_noiseStrength, sizeof( _noiseStrength ), 0, hipMemcpyHostToDevice ) );

	// Clear cache
	clearCache();
}
